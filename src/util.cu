/* util.cu

   CUDA utilities. */


#include <hip/hip_runtime.h>
#include <cassert>

extern "C" {

size_t cuMemAvail() {
  size_t free, total;
  assert(hipSuccess == hipMemGetInfo(&free, &total));
  return free;
}

void* cuMalloc(size_t size) {
  void* p;
  assert(hipSuccess == hipMalloc(&p, size));
  return p;
}

void cuFree(void* p) { assert(hipSuccess == hipFree(p)); }

void cuClear(void* p, size_t size) {
  assert(hipSuccess == hipMemset(p, 0, size));
}

void cuUpload(void* devDst, const void* hostSrc, size_t size) {
  assert(hipSuccess == hipMemcpy(devDst, hostSrc, size,
                                   hipMemcpyHostToDevice));
}

void cuDownload(void* hostDst, const void* devSrc, size_t size) {
  assert(hipSuccess == hipMemcpy(hostDst, devSrc, size,
         hipMemcpyDeviceToHost));
}

void cuPin(void* p, size_t size) {
  assert(hipSuccess == hipHostRegister(p, size, hipHostRegisterPortable));
}

void cuUnpin(void* p) {
  assert(hipSuccess == hipHostUnregister(p));
}

} // end extern "C"
