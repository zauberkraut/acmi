/* util.cu

   CUDA utilities. */

#include "acmi.h"

extern "C" {

void cuCheck(const char* fname, const size_t lnum) {
  auto r = hipPeekAtLastError();
  if (hipSuccess != r) {
    fatal("%s line %d: %s", fname, lnum, hipGetErrorString(r));
  }
}

size_t cuMemAvail() {
  size_t free, total;
  hipMemGetInfo(&free, &total);
  CUCHECK;
  return free;
}

void* cuMalloc(size_t size) {
  void* p;
  hipMalloc(&p, size);
  CUCHECK;
  return p;
}

void cuFree(void* p) {
  hipFree(p);
  CUCHECK;
}

void cuClear(void* p, size_t size) {
  hipMemset(p, 0, size);
  CUCHECK;
}

void cuUpload(void* devDst, const void* hostSrc, size_t size) {
  hipMemcpy(devDst, hostSrc, size, hipMemcpyHostToDevice);
  CUCHECK;
}

void cuDownload(void* hostDst, const void* devSrc, size_t size) {
  hipMemcpy(hostDst, devSrc, size, hipMemcpyDeviceToHost);
  CUCHECK;
}

void cuPin(void* p, size_t size) {
  hipHostRegister(p, size, hipHostRegisterPortable);
  CUCHECK;
}

void cuUnpin(void* p) {
  hipHostUnregister(p);
  CUCHECK;
}

} // end extern "C"
