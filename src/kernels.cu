#include "hip/hip_runtime.h"
/* kernels.cu

   Custom ACMI CUDA kernels. */

#include "acmi.h"

namespace {

enum { SUM_SWEEP_FACTOR = 4 };

/* Kernel parameters. */
int g_maxBlocksPerGrid, g_maxThreadsPerBlock,
    g_blocksPerKernel,  g_threadsPerBlock,
    g_threadsPerKernel;

/* Copies elements from one nxn matrix to another, converting them to the
   precision of the destination matrix. */
template<typename T, typename U> __global__ void
kernCopy(T* dst, const U* src, const int64_t n2) {
  const int offset = blockIdx.x*blockDim.x + threadIdx.x;
  const int stride = gridDim.x*blockDim.x;
  const T* end = dst + n2;
  src += offset;
  dst += offset;

  for (; dst < end; dst += stride, src += stride) {
    *dst = *src;
  }
}

template<typename T> __global__ void
kernAddId(T* a, const T alpha, const int n) {
  const T* end = a + n*n;
  a += (blockIdx.x*blockDim.x + threadIdx.x)*(n + 1);
  const int stride = gridDim.x*blockDim.x*(n + 1);

  for (; a < end; a += stride) {
    *a += alpha;
  }
}

} // end anonymous namespace

extern "C" {

/* Sets up kernel parameters. */
void cuSetUp(const int maxBlocksPerKernel, const int n) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0); // assumes usage of the first device
  debug("setting up kernels on %s", prop.name);
  g_maxBlocksPerGrid = prop.maxGridSize[0];
  g_maxThreadsPerBlock = prop.maxThreadsPerBlock;
  const int64_t n2 = n*n;
  g_blocksPerKernel = (n2 + g_maxThreadsPerBlock - 1) / g_maxThreadsPerBlock;
  g_blocksPerKernel = iMin(maxBlocksPerKernel, g_blocksPerKernel);
  g_threadsPerBlock = iMin(g_maxThreadsPerBlock, n2);
  g_threadsPerKernel = g_threadsPerBlock*g_blocksPerKernel;
  debug("max  blocks/grid  : %d\n"
        "max threads/block : %d\n"
        "     blocks/kernel: %d\n"
        "    threads/block : %d\n"
        "    threads/kernel: %d", g_maxBlocksPerGrid, g_maxThreadsPerBlock,
        g_blocksPerKernel, g_threadsPerBlock, g_threadsPerKernel);
}

void cuShutDown() {
  debug("shutting down kernels");
}

/* Doubles matrix precision. */
void cuPromote(void* dst, void* src, int srcElemSize, int64_t n2) {
  kernCopy<<<g_blocksPerKernel, g_threadsPerBlock>>>
    ((double*)dst, (const float*)src, n2);
}

/* Adds alpha*I to the nxn matrix backed by the device array elems. */
void cuAddId(void* elems, double alpha, int n, int elemSize) {
  const int nThreads = iMin(g_maxThreadsPerBlock, n);
  switch (elemSize) {
  case 4: kernAddId<<<1, nThreads>>>((float*)elems, (float)alpha, n); break;
  case 8: kernAddId<<<1, nThreads>>>((double*)elems, alpha, n);       break;
  }
}

} // end extern "C"
