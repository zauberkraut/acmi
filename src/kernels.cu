/* kernels.cu

   Custom ACMI CUDA kernels. */


#include <hip/hip_runtime.h>
#include <assert.h>

extern "C" void* cuMalloc(size_t size) {
  void* p;
  assert(hipMalloc(&p, size) == hipSuccess);
  return p;
}

extern "C" void cuFree(void* p) { assert(hipFree(p) == hipSuccess); }

extern "C" void cuClear(void* p, size_t size) {
  assert(hipMemset(p, 0, size) == hipSuccess);
}

extern "C" void cuUpload(void* devDst, const void* hostSrc, size_t size) {
  assert(hipMemcpy(devDst, hostSrc, size, hipMemcpyHostToDevice) ==
         hipSuccess);
}

extern "C" void cuDownload(void* hostDst, const void* devSrc, size_t size) {
  assert(hipMemcpy(hostDst, devSrc, size, hipMemcpyDeviceToHost) ==
         hipSuccess);
}
