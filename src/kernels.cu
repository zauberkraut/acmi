#include "hip/hip_runtime.h"
/* kernels.cu

   Custom ACMI CUDA kernels. */

#include <cassert>
#include <hip/hip_fp16.h>
#include <stdint.h>

extern "C" {

static __global__ void kern16to32(float* dst, const __half* src,
                                  const int64_t n2) {
  for (int64_t i = 0; i < n2; i++) {
    dst[i] = __half2float(src[i]);
  }
}

static __global__ void kern32to64(double* dst, const float* src,
                                  const int64_t n2) {
  for (int64_t i = 0; i < n2; i++) {
    dst[i] = src[i];
  }
}

void cuPromote(void* dst, void* src, int srcElemSize, int64_t n2) {
  switch (srcElemSize) {
  case 2: kern16to32<<<1, 1>>>((float*)dst, (const __half*)src, n2); break;
  case 4: kern32to64<<<1, 1>>>((double*)dst, (const float*)src, n2); break;
  case 8: /* WIP */; break;
  }
  assert(hipSuccess == hipGetLastError());
}

static __global__ void kern16SetDiag(__half* elems, float alpha, int n) {
  __half a = __float2half(alpha);
  for (int i = 0; i < n; i++) {
    elems[i*n + i] = a;
  }
}

static __global__ void kern32SetDiag(float* elems, float alpha, int n) {
  for (int i = 0; i < n; i++) {
    elems[i*n + i] = alpha;
  }
}

static __global__ void kern64SetDiag(double* elems, double alpha, int n) {
  for (int i = 0; i < n; i++) {
    elems[i*n + i] = alpha;
  }
}

void cuSetDiag(void* elems, double alpha, int n, int elemSize) {
  switch (elemSize) {
  case 2:
    kern16SetDiag<<<1, 1>>>((__half*)elems, alpha, n);
    break;
  case 4: kern32SetDiag<<<1, 1>>>((float*)elems, alpha, n);  break;
  case 8: kern64SetDiag<<<1, 1>>>((double*)elems, alpha, n); break;
  }
  assert(hipSuccess == hipGetLastError());
}

static __global__ void kern16AddDiag(__half* a, const float alpha,
                                     const int n) {
  for (int i = 0; i < n; i++) {
    const int j = i*n + i;
    a[j] = __float2half(__half2float(a[j]) + alpha);
  }
}

static __global__ void kern32AddDiag(float* a, const float alpha,
                                     const int n) {
  for (int i = 0; i < n; i++) {
    const int j = i*n + i;
    a[j] = a[j] + alpha;
  }
}

static __global__ void kern64AddDiag(double* a, const double alpha,
                                     const int n) {
  for (int i = 0; i < n; i++) {
    const int j = i*n + i;
    a[j] = a[j] + alpha;
  }
}

void cuAddDiag(void* elems, double alpha, int n, int elemSize) {
  switch (elemSize) {
  case 2: kern16AddDiag<<<1, 1>>>((__half*)elems, alpha, n); break;
  case 4: kern32AddDiag<<<1, 1>>>((float*)elems,  alpha, n); break;
  case 8: kern64AddDiag<<<1, 1>>>((double*)elems, alpha, n); break;
  }
  assert(hipSuccess == hipGetLastError());
}

__device__ double d_froNorm;

static __global__ void kern16Norm(const __half* a, const int64_t n2) {
  double sum = 0;
  for (int64_t i = 0; i < n2; i++) {
    double e = __half2float(a[i]);
    sum += e*e;
  }
  d_froNorm = sqrt(sum);
}

static __global__ void kern32Norm(const float* a, const int64_t n2) {
  double sum = 0;
  for (int64_t i = 0; i < n2; i++) {
    double e = a[i];
    sum += e*e;
  }
  d_froNorm = sqrt(sum);
}

static __global__ void kern64Norm(const double* a, const int64_t n2) {
  double sum = 0;
  for (int64_t i = 0; i < n2; i++) {
    double e = a[i];
    sum += e*e;
  }
  d_froNorm = sqrt(sum);
}

double cuNorm(void* elems, int64_t n2, int elemSize) {
  switch (elemSize) {
  case 2: kern16Norm<<<1, 1>>>((__half*)elems, n2); break;
  case 4: kern32Norm<<<1, 1>>>((float*)elems, n2);  break;
  case 8: kern64Norm<<<1, 1>>>((double*)elems, n2); break;
  }

  assert(hipSuccess == hipGetLastError());
  typeof(d_froNorm) froNorm;
  hipMemcpyFromSymbol(&froNorm, HIP_SYMBOL(d_froNorm), sizeof(froNorm), 0, hipMemcpyDeviceToHost);
  return froNorm;
}

static __global__ void kern16NormSubFromI(__half* a, int n) {
  double sum = 0;
  for (int col = 0; col < n; col++) {
    for (int row = 0; row < n; row++) {
      int i = col*n + row;
      double e = (col == row) - __half2float(a[i]);
      sum += e*e;
    }
  }
  d_froNorm = sqrt(sum);
}

static __global__ void kern32NormSubFromI(float* a, int n) {
  double sum = 0;
  for (int col = 0; col < n; col++) {
    for (int row = 0; row < n; row++) {
      int i = col*n + row;
      double e = (col == row) - a[i];
      sum += e*e;
    }
  }
  d_froNorm = sqrt(sum);
}

static __global__ void kern64NormSubFromI(double* a, int n) {
  double sum = 0;
  for (int col = 0; col < n; col++) {
    for (int row = 0; row < n; row++) {
      int i = col*n + row;
      double e = (col == row) - a[i];
      sum += e*e;
    }
  }
  d_froNorm = sqrt(sum);
}

double cuNormSubFromI(void* elems, int n, int elemSize) {
  switch (elemSize) {
  case 2: kern16NormSubFromI<<<1, 1>>>((__half*)elems, n); break;
  case 4: kern32NormSubFromI<<<1, 1>>>((float*)elems, n);  break;
  case 8: kern64NormSubFromI<<<1, 1>>>((double*)elems, n); break;
  }

  assert(hipSuccess == hipGetLastError());
  typeof(d_froNorm) froNorm;
  hipMemcpyFromSymbol(&froNorm, HIP_SYMBOL(d_froNorm), sizeof(froNorm), 0, hipMemcpyDeviceToHost);
  return froNorm;
}

static __global__ void kernHgeam(float alpha, __half* a, float beta, __half* b,
                                 __half* c, int64_t n2) {
  for (int64_t i = 0; i < n2; i++) {
    c[i] = __float2half(alpha * __half2float(a[i]) + beta * __half2float(b[i]));
  }
}

void cuHgeam(float alpha, void* a, float beta, void* b, void* c, int64_t n2) {
  kernHgeam<<<1, 1>>>(alpha, (__half*)a, beta, (__half*)b, (__half*)c, n2);
  assert(hipSuccess == hipGetLastError());
}

} // end extern "C"
